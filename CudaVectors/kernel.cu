﻿#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addVectors(int* a, int* b, int* c) {
    int i = threadIdx.x;

    c[i] = a[i] + b[i];
}

int main()
{
    int a[] = { 1,2,3 };
    int b[] = { 4,5,6 };
    int c[sizeof(a) / sizeof(int)] = { 0 };

    int* cudaA = 0;
    int* cudaB = 0;
    int* cudaC = 0;

    // allocate gpu memory
    hipMalloc(&cudaA, sizeof(a));
    hipMalloc(&cudaB, sizeof(b));
    hipMalloc(&cudaC, sizeof(c));

    // copy vectors to gpu memory
    hipMemcpy(cudaA, a, sizeof(a), hipMemcpyHostToDevice);
    hipMemcpy(cudaB, b, sizeof(b), hipMemcpyHostToDevice);

    // run addVectors function with grid of 1 block and pass parameters
    addVectors <<< 1, sizeof(a) / sizeof(int) >>> (cudaA, cudaB, cudaC);

    hipMemcpy(c, cudaC, sizeof(c), hipMemcpyDeviceToHost);

    return 0;
}
